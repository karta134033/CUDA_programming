#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 32
#define GROUP_OF_PIXELS 1

__global__ void mandelKernel(float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_OF_PIXELS;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    for (int i = 0; i < GROUP_OF_PIXELS && col < resX; i++, col++) {
        float x = lowerX + col * stepX;
        float y = lowerY + row * stepY;
        float z_re = x, z_im = y;
        int val = 0;
        for (; val < maxIterations; ++val) {
            if (z_re * z_re + z_im * z_im > 4.f) break;
            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = x + new_re;
            z_im = y + new_im;
        }
        img[row * resX + col] = val;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations) {
    int *temp_img; 
    int *output;
    size_t pitch;
    hipHostAlloc((void **)&output, resX * resY * sizeof(int), hipHostMallocDefault);
    hipMallocPitch((void **) &temp_img, &pitch, resX * sizeof (int), resY);
    
    int group_block_size = BLOCK_SIZE * GROUP_OF_PIXELS;
    static int x_blocks = resX % group_block_size == 0 ? resX / group_block_size : resX / group_block_size + 1;
    static int y_blocks = resY % BLOCK_SIZE == 0 ? resY / BLOCK_SIZE : resY / BLOCK_SIZE + 1;
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 num_block(x_blocks, y_blocks);
    mandelKernel<<<num_block, block_size>>>(upperX, upperY, lowerX, lowerY, temp_img, resX, resY, maxIterations);

    hipMemcpy(output, temp_img, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    memcpy(img, output, resX * resY * sizeof(int));
    hipHostFree(output);
    hipFree(temp_img);
}